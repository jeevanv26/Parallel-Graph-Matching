/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2021                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	/** YOUR CODE GOES BELOW **/
  int numThreads = gridDim.x * blockDim.x;
  int tid = (blockDim.x * blockIdx.x ) + threadIdx.x;
  if(numThreads > numNodes){
    if(tid < numNodes){
      if(strongNeighbor[strongNeighbor[tid]] == tid && matches[tid] == -2)
        matches[tid] = strongNeighbor[tid];
    }
  }
  else{
      for(int x = tid; x < numNodes; x += numThreads ){
        if(strongNeighbor[strongNeighbor[x]] == x && matches[x] == -2)
          matches[x] = strongNeighbor[x];
      }
}
	/** YOUR CODE GOES ABOVE **/
}
