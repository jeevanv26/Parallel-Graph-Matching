/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2021                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void markFilterEdges_gpu(int * src, int * dst, int * matches, int * keepEdges, int numEdges) {
	/** YOUR CODE GOES BELOW **/
  int numThreads = gridDim.x * blockDim.x;
  int tid = (blockDim.x * blockIdx.x ) + threadIdx.x;
  if(numThreads > numEdges){
    if(tid < numEdges){
        keepEdges[tid] = 1;
        int source = src[tid];
        int destination = dst[tid];
        if( matches[source] != -2 || matches[destination] != -2)
          keepEdges[tid] = 0;
      }
  }
  else{
    for(int x = tid; x < numEdges; x += numThreads ){
        keepEdges[x] = 1;
        int source = src[x];
        int destination = dst[x];
        if( matches[source] != -2 || matches[destination] != -2)
          keepEdges[x] = 0;

      }
  }

	/** YOUR CODE GOES ABOVE **/
}
