/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2021                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void exclusive_prefix_sum_gpu(int * oldSum, int * newSum, int distance, int numElements) {
	/** YOUR CODE GOES BELOW **/
  int numThreads = gridDim.x * blockDim.x;
  int tid = (blockDim.x * blockIdx.x ) + threadIdx.x;
    if(numThreads > numElements){
      if(tid < numElements){
        if( distance == 0){
          newSum[tid] = 0;
          if(tid-1 >= 0)
          newSum[tid] = oldSum[tid-1];
        }
        else{
          newSum[tid] = oldSum[tid];
          if(tid - distance >= 0)
          newSum[tid] = oldSum[tid-distance] + oldSum[tid];
        }
      }
    }
  else{
    if( distance == 0){
      for(int x = tid; x < numElements; x += numThreads ){
          newSum[x] = 0;
          if(x-1 >= 0)
            newSum[x] = oldSum[x-1];
      }
    }
    else{
      for(int x = tid; x < numElements; x += numThreads ){
          newSum[x] = oldSum[x];
        if(x - distance >= 0)
          newSum[x] = oldSum[x-distance] + oldSum[x];
      }
  }
}
	/** YOUR CODE GOES ABOVE **/

}
